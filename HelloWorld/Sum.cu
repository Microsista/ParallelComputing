
#include <hip/hip_runtime.h>


using namespace std;

__global__ void sum(unsigned char* d_color, unsigned char* d_colorBlurred, int numRows, int numCols, int blurRadius) {
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	int color = 0;
	for (int i = -blurRadius; i <= blurRadius; i++) {
		for (int j = -blurRadius; j <= blurRadius; j++) {
			int2 sample2DPos = make_int2(blockIdx.x * blockDim.x + threadIdx.x + j,
				blockIdx.y * blockDim.y + threadIdx.y + i);

			int sample1DPos = sample2DPos.y * numCols + sample2DPos.x;
			color += d_color[sample1DPos];
		}
	}
	color /= (2 * blurRadius + 1) * (2 * blurRadius + 1);


	d_colorBlurred[thread_1D_pos] = (unsigned char)color;


	/*int px = blockIdx.x * blockDim.x + threadIdx.x;
	int py = blockIdx.y * blockDim.y + threadIdx.y;
	if (px >= numCols || py >= numRows) {
		return;
	}
	int i = py * numCols + px;
	redChannel[i] = inputImageRGB[i].x;
	greenChannel[i] = inputImageRGB[i].y;
	blueChannel[i] = inputImageRGB[i].z;*/



	/*__shared__ int shared[numberOfThreads + 2 * blurRadius];
	int globalIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int localIndex = threadIdx.x + blurRadius;

	if (globalIndex < workSize) {

		auto loadToShared = [&] {
			auto actual = [&] {
				shared[localIndex] = input[globalIndex];
			};

			auto halo = [&] {
				if (threadIdx.x < blurRadius) {
					shared[localIndex - blurRadius] = globalIndex >= blurRadius ? input[globalIndex - blurRadius] : 0;
					shared[localIndex + numberOfThreads] = globalIndex < (workSize - numberOfThreads) ? input[globalIndex + numberOfThreads] : 0;
				}
			};

			actual();
			halo();
			__syncthreads();
		};

		auto sumNeighbouringValues = [&] {
			int result = 0;
			for (int offset = -blurRadius; offset <= blurRadius; offset++) {
				result += shared[localIndex + offset];
			}
			return result;
		};

		loadToShared();
		output[globalIndex] = sumNeighbouringValues();
	}*/
}