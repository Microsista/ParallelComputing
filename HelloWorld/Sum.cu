#include "hip/hip_runtime.h"
#include "Globals.cuh"
#include "Utilities.cuh"
#include <>

#include <algorithm>

using namespace std;

__global__ void sum(Uchar3* input, Uchar3* output, int workSize, int width) {
	struct Uchar3 {
		unsigned char x, y, z;
		Uchar3() {}
		Uchar3(unsigned char x, unsigned char y, unsigned char z) : x{ x }, y{ y }, z{ z } {}
		Uchar3 operator*(const Uchar3& rhs) {
			return Uchar3(this->x * rhs.x, this->y * rhs.y, this->z * rhs.z);
		}
		Uchar3 operator*(float rhs) {
			return Uchar3(this->x * rhs, this->y * rhs, this->z * rhs);
		}
		Uchar3 operator+(const Uchar3& rhs) {
			return Uchar3(this->x + rhs.x, this->y + rhs.y, this->z + rhs.z);
		}
	};

	int globalIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int globalX = globalIndex % width;
	int globalY = globalIndex / width;

	Uchar3 result;
	/*for (int offset = -blurRadius; offset <= blurRadius; offset++) {
		result += input[min(workSize, max(0, globalIndex + offset))];
	}*/

	float weights[] = {
		0.05f, 0.09f, 0.12f, 0.15f, 0.16f, 0.15f, 0.12f, 0.09f, 0.05f
	};
	for (int i = -blurRadius; i <= blurRadius; i++) {
		for (int j = -blurRadius; j <= blurRadius; j++) {
			int x = j, y = i;
			if (globalIndex % width + j < 0)
				x = 0;
			if (globalIndex % width + j > width)
				x = 0;
			if (globalIndex / width + i < 0)
				y = 0;
			if (globalIndex / width + i > workSize/width)
				y = 0;

			result = result + *input[globalX + x + (width * (globalY + y))] * weights[j + blurRadius] * weights[i + blurRadius];
		}
	}

	*output[globalIndex] = result;
}