
#include <hip/hip_runtime.h>


using namespace std;

__global__ void sum(unsigned char* d_color, unsigned char* d_colorBlurred, int numRows, int numCols, int blurRadius) {
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	float weights[] = {
		0.05f, 0.09f, 0.12f, 0.15f, 0.16f, 0.15f, 0.12f, 0.09f, 0.05f
	};
	int color = 0;
	for (int i = -blurRadius; i <= blurRadius; i++) {
		for (int j = -blurRadius; j <= blurRadius; j++) {
			int x = j, y = i;
			if (thread_2D_pos.x + j <= 0)
				x = 0;
			if (thread_2D_pos.x + j > numCols)
				x = 0;
			if (thread_2D_pos.y + i <= 0)
				y = 0;
			if (thread_2D_pos.y + i > numRows)
				y = 0;

			int2 sample2DPos = make_int2(blockIdx.x * blockDim.x + threadIdx.x + x,
				blockIdx.y * blockDim.y + threadIdx.y + y);

			int sample1DPos = sample2DPos.y * numCols + sample2DPos.x;
			
			color += d_color[sample1DPos];
		}
	}
	color /= (2 * blurRadius + 1) * (2 * blurRadius + 1);


	d_colorBlurred[thread_1D_pos] = (unsigned char)color;


	/*int px = blockIdx.x * blockDim.x + threadIdx.x;
	int py = blockIdx.y * blockDim.y + threadIdx.y;
	if (px >= numCols || py >= numRows) {
		return;
	}
	int i = py * numCols + px;
	redChannel[i] = inputImageRGB[i].x;
	greenChannel[i] = inputImageRGB[i].y;
	blueChannel[i] = inputImageRGB[i].z;*/



	/*__shared__ int shared[numberOfThreads + 2 * blurRadius];
	int globalIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int localIndex = threadIdx.x + blurRadius;

	if (globalIndex < workSize) {

		auto loadToShared = [&] {
			auto actual = [&] {
				shared[localIndex] = input[globalIndex];
			};

			auto halo = [&] {
				if (threadIdx.x < blurRadius) {
					shared[localIndex - blurRadius] = globalIndex >= blurRadius ? input[globalIndex - blurRadius] : 0;
					shared[localIndex + numberOfThreads] = globalIndex < (workSize - numberOfThreads) ? input[globalIndex + numberOfThreads] : 0;
				}
			};

			actual();
			halo();
			__syncthreads();
		};

		auto sumNeighbouringValues = [&] {
			int result = 0;
			for (int offset = -blurRadius; offset <= blurRadius; offset++) {
				result += shared[localIndex + offset];
			}
			return result;
		};

		loadToShared();
		output[globalIndex] = sumNeighbouringValues();
	}*/
}