#include "hip/hip_runtime.h"
#include "Globals.cuh"

#include <>

__global__ void sum(int* input, int* output, int workSize) {
	__shared__ int shared[numberOfThreads + 2 * blurRadius];
	int globalIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int localIndex = threadIdx.x + blurRadius;
	
	if (globalIndex < workSize) {
		auto loadToShared = [&] {
			auto actual = [&] {
				shared[localIndex] = input[globalIndex];
			};

			auto halo = [&] {
				if (threadIdx.x < blurRadius) {
					shared[localIndex - blurRadius] = globalIndex >= blurRadius ? input[globalIndex - blurRadius] : 0;
					shared[localIndex + numberOfThreads] = globalIndex < (workSize - numberOfThreads) ? input[globalIndex + numberOfThreads] : 0;
				}
			};

			actual();
			halo();
			__syncthreads(); 
		};

		auto sumNeighbouringValues = [&] {
			int result = 0;
			for (int offset = -blurRadius; offset <= blurRadius; offset++) {
				result += shared[localIndex + offset];
			}
			return result;
		};
		
		loadToShared();
		output[globalIndex] = sumNeighbouringValues();
	}
}